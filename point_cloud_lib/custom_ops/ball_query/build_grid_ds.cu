#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////
/// \file build_grid_ds.cu
///
/// \brief Implementation of the CUDA operations to build the data structure to 
///     access the sparse regular grid. 
///
/// \copyright Copyright (c) 2023 Pedro Hermosilla, TU-Wien, Austria  
///            See the LICENSE file at the top-level directory of 
///            this distribution.
///
/// \author pedro hermosilla (phermosilla@cvl.tuwien.ac.at)
/////////////////////////////////////////////////////////////////////////////

#include "ball_query_utils.cuh"
#include "math_helper.cuh"
#include "grid_utils.cuh"
#include "../utils.cuh"

#include "build_grid_ds.cuh"

///////////////////////// GPU
 
/**
 *  GPU kernel to compute the grid data structure.
 *  @param  pNumPts         Number of points.
 *  @param  pKeys           Array of keys.
 *  @param  pNumCells       Number of cells.
 *  @param  pOutDS          Output array with the data structure.
 *  @paramT D                       Number of dimensions.
 */
 template<int D>
 __global__ void build_grid_gpu_kernel(
    const unsigned int pNumPts,
    const int64_m* __restrict__ pKeys,
    const int* __restrict__ pNumCells,
    int2* __restrict__ pOutDS)
{
    const ipoint<D>* numCellsNewPtr = (const ipoint<D>*)pNumCells;

    int initPtIndex = compute_global_index_gpu_funct();
    int totalThreads = compute_total_threads_gpu_funct();

    for(int curPtIndex = initPtIndex; curPtIndex < pNumPts; curPtIndex += totalThreads)
    {
        //Get the key and compute the index into the ds.
        int64_m curKey = pKeys[curPtIndex];
        int dsIndex = compute_ds_index_from_key_gpu_funct(curKey, numCellsNewPtr[0]);
        
        //Check if it is the first point in the ds cell.
        int prevPtIndex = curPtIndex-1;
        if(prevPtIndex >= 0){
            if(dsIndex != 
                compute_ds_index_from_key_gpu_funct(pKeys[prevPtIndex], numCellsNewPtr[0])){
                    pOutDS[dsIndex].x = curPtIndex;
            }
        }

        //Check if it is the last point in the ds cell.
        int nextPtIndex = curPtIndex+1;
        if(nextPtIndex == pNumPts){
            pOutDS[dsIndex].y = pNumPts;
        }else if(dsIndex != 
            compute_ds_index_from_key_gpu_funct(pKeys[nextPtIndex], numCellsNewPtr[0])){
            pOutDS[dsIndex].y = nextPtIndex;
        }
    }
}

///////////////////////// CPU

torch::Tensor build_grid_ds(
    torch::Tensor pKeys,
    torch::Tensor pGridSize,
    std::vector<int64_t>& pOutShape) {

    // Get the number of dimensions and points.
    int numDims = pGridSize.size(0);
    int numPts = pKeys.size(0);

    // Get device properties.
    hipDeviceProp_t props = get_cuda_device_properties();

    // Get the function pointer.
    void* funcPtr = nullptr;
    DIMENSION_FUNCT_PTR(numDims, build_grid_gpu_kernel, funcPtr);

    // Calculate the ideal number of blocks for the selected block size.
    unsigned int numMP = props.multiProcessorCount;
    unsigned int blockSize = props.warpSize*2;
    unsigned int numBlocks = get_max_active_block_x_sm(
        blockSize, funcPtr, 0);

    // Calculate the total number of blocks to execute.
    unsigned int execBlocks = numPts/blockSize;
    execBlocks += (numPts%blockSize != 0)?1:0;
    unsigned int totalNumBlocks = numMP*numBlocks;
    totalNumBlocks = (totalNumBlocks > execBlocks)?execBlocks:totalNumBlocks;

    // Create output.
    auto tensorOptions = torch::TensorOptions().dtype(torch::kInt32).
        device(pKeys.device().type(), pKeys.device().index());
    auto outTensor = torch::zeros(pOutShape, tensorOptions);

    // Call the cuda kernel.
    DIMENSION_SWITCH_CALL(numDims, build_grid_gpu_kernel, totalNumBlocks, blockSize, 0,
        numPts, 
        (const int64_m*)pKeys.data_ptr(), 
        (const int*)pGridSize.data_ptr(), 
        (int2*)outTensor.data_ptr());

    // Return result.
    return outTensor;
}
